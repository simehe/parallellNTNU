#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <time.h>

#define dT 0.2f
#define G 0.6f
#define BLOCK_SIZE 128

// Global variables
int num_planets;
int num_timesteps;

// Host arrays
float2* velocities;
float4* planets;

// Device arrays 
float2* velocities_d;
float4* planets_d;


// Parse command line arguments
void parse_args(int argc, char** argv){
    if(argc != 2){
        printf("Useage: nbody num_timesteps\n");
        exit(-1);
    }
    
    num_timesteps = strtol(argv[1], 0, 10);
}

// Reads planets from planets.txt
void read_planets(){

    FILE* file = fopen("planets4096.txt", "r");
    if(file == NULL){
        printf("'planets.txt' not found. Exiting\n");
        exit(-1);
    }

    char line[200];
    fgets(line, 200, file);
    sscanf(line, "%d", &num_planets);

    planets = (float4*)malloc(sizeof(float4)*num_planets);
    velocities = (float2*)malloc(sizeof(float2)*num_planets);

    for(int p = 0; p < num_planets; p++){
        fgets(line, 200, file);
        sscanf(line, "%f %f %f %f %f",
                &planets[p].x,
                &planets[p].y,
                &velocities[p].x,
                &velocities[p].y,
                &planets[p].z);
    }

    fclose(file);
}

// Writes planets to file
void write_planets(int timestep){
    char name[20];
    int n = sprintf(name, "planets_out.txt");

    FILE* file = fopen(name, "wr+");

    for(int p = 0; p < num_planets; p++){
        fprintf(file, "%f %f %f %f %f\n",
                planets[p].x,
                planets[p].y,
                velocities[p].x,
                velocities[p].y,
                planets[p].z);
    }

    fclose(file);
}

// TODO 7. Calculate the change in velocity for p, caused by the interaction with q
__device__ float2 calculate_velocity_change_planet(float4 p, float4 q){
    float2 r;
    r.x = q.x - p.x;
    r.y = q.y - p.y;
    if(r.x == 0.0 && r.y == 0.0){
        float2 v = {0.0f, 0.0f};
        return v;
    }    
    float abs_dist = sqrt(r.x*r.x + r.y*r.y);
    float dist_cubed = abs_dist*abs_dist*abs_dist;
    float2 dv;
    dv.x = dT*G*q.z/dist_cubed * r.x;
    dv.y = dT*G*q.z/dist_cubed * r.y;
    return dv;



}

// TODO 5. Calculate the change in velocity for my_planet, caused by the interactions with a block of planets
__device__ float2 calculate_velocity_change_block(float4 my_planet, float4* shared_planets){
    float2 velocity = {0.0f, 0.0f};
    for(int i = 0; i < blockDim.x; i++){
        float2 tempv = calculate_velocity_change_planet(my_planet, shared_planets[i]);
        velocity.x += tempv.x;
        velocity.y += tempv.y;

    }    
    return velocity;


}

// TODO 4. Update the velocities by calculating the planet interactions
__global__ void update_velocities(float4* planets, float2* velocities, int num_planets){
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    float4 my_planet = planets[thread_id];
    __shared__ float4 shared_planets[BLOCK_SIZE];
    for(int i = 0; i < num_planets; i+=blockDim.x){
        shared_planets[threadIdx.x] = planets[i + threadIdx.x];
        __syncthreads();
        float2 tempv = calculate_velocity_change_block(my_planet, shared_planets);
        velocities[thread_id].x += tempv.x;
        velocities[thread_id].y += tempv.y;
        __syncthreads();
    }    


}

// TODO 7. Update the positions of the planets using the new velocities
__global__ void update_positions(float4* planets, float2* velocities, int num_planets){
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    planets[thread_id].x += velocities[thread_id].x * dT;
    planets[thread_id].y += velocities[thread_id].y * dT;

}


int main(int argc, char** argv){

    parse_args(argc, argv);
    read_planets();

    // TODO 1. Allocate device memory, and transfer data to device 
    hipMalloc(&planets_d, sizeof(float4)*num_planets);
    hipMalloc(&velocities_d, sizeof(float2)*num_planets);
    hipMemcpy(planets_d, planets, sizeof(float4)*num_planets, hipMemcpyHostToDevice);
    hipMemcpy(velocities_d, velocities, sizeof(float2)*num_planets, hipMemcpyHostToDevice);


    // Calculating the number of blocks
    int num_blocks = num_planets/BLOCK_SIZE + ((num_planets%BLOCK_SIZE == 0) ? 0 : 1);
    double start_t = clock();		

    // Main loop
    for(int t = 0; t < num_timesteps; t++){
        // TODO 2. Call kernels
        update_velocities<<<num_blocks,BLOCK_SIZE >>>(planets_d, velocities_d, num_planets);
        update_positions<<<num_blocks, BLOCK_SIZE>>>(planets_d, velocities_d, num_planets);
    }
    double end_t = clock();
    // TODO 3. Transfer data back to host
    hipMemcpy(velocities, velocities_d, sizeof(float2)*num_planets, hipMemcpyDeviceToHost);
    hipMemcpy(planets, planets_d, sizeof(float4)*num_planets, hipMemcpyDeviceToHost);
    double total_t = (double)(end_t - start_t);
    hipDeviceSynchronize();
    printf("Time taken to run: %f ", total_t/CLOCKS_PER_SEC); 
    // Output
    write_planets(num_timesteps);
    
    hipFree(planets_d);
    hipFree(velocities_d);
    free(planets);
    free(velocities);	
}
